#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include <fstream>
#include <cstdint>
#include <time.h>
#include <ctime>

using namespace std;

typedef struct  WAV_HEADER
{
	/* RIFF Chunk Descriptor */
	uint8_t         RIFF[4];        // RIFF Header Magic header
	uint32_t        ChunkSize;      // RIFF Chunk Size
	uint8_t         WAVE[4];        // WAVE Header
	/* "fmt" sub-chunk */
	uint8_t         fmt[4];         // FMT header
	uint32_t        Subchunk1Size;  // Size of the fmt chunk
	uint16_t        AudioFormat;    // Audio format 1=PCM,6=mulaw,7=alaw,     257=IBM Mu-Law, 258=IBM A-Law, 259=ADPCM
	uint16_t        NumOfChan;      // Number of channels 1=Mono 2=Sterio
	uint32_t        SamplesPerSec;  // Sampling Frequency in Hz
	uint32_t        bytesPerSec;    // bytes per second
	uint16_t        blockAlign;     // 2=16-bit mono, 4=16-bit stereo
	uint16_t        bitsPerSample;  // Number of bits per sample
	/* "data" sub-chunk */
	uint8_t         Subchunk2ID[4]; // "data"  string
	uint32_t        Subchunk2Size;  // Sampled data length
} wav_hdr;

// Function prototypes
int getFileSize(FILE* inFile);
double* filter(wav_hdr wavHeader, double limit_freq);

__global__ void filterr(int8_t* buffer_d, int8_t* buffer_dd, double* filtr)				//a gpu function that calculates the convolution
{																						//for each sample represented by each thread
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	double suma = 0;
	if (i > 30)
	{
		for (int j = 0; j < 31; j++)
			suma += filtr[j] * abs(buffer_d[i - j]);
	}
	else 
		for (int k = 0; k < i; k++)
			suma += filtr[k] * abs(buffer_d[i - k]);
	
	buffer_dd[i] = suma;
}

int main(int argc, char* argv[])
{
	wav_hdr wavHeader;
	wav_hdr* wavHeader_d;
	wav_hdr* wavHeaderPtr = &wavHeader;
	wav_hdr* wavHeaderPtr1 = &wavHeader;
	int headerSize = sizeof(wav_hdr);
	int filelength = 0;
	const char* filePath;
	string input;
	if (argc <= 1)
	{
		cout << "Input wave file name: ";
		cin >> input;
		cin.get();
		filePath = input.c_str();
	}
	else
	{
		filePath = argv[1];
		cout << "Input wave file name: " << filePath << endl;
	}

	FILE* wavFile = fopen(filePath, "r");
	FILE* wavFile_d = fopen(filePath, "r");
	FILE* output = fopen("output.wav", "w");
	if (wavFile == nullptr)
	{
		fprintf(stderr, "Unable to open wave file: %s\n", filePath);
		return 1;
	}

	//Read the header
	size_t bytesRead = fread(&wavHeader, 1, headerSize, wavFile);
	size_t bytesWritten = fwrite(wavHeaderPtr1, sizeof(wav_hdr), 1, output);

	static const uint64_t BUFFER_SIZE = wavHeader.Subchunk2Size;
	/*cudaMalloc((void**)&wavHeader_d, sizeof(wav_hdr));
	cudaMemcpy(wavHeader_d, wavHeaderPtr, BUFFER_SIZE * sizeof(int8_t), cudaMemcpyHostToDevice);*/

	cout << "Header Read " << bytesRead << " bytes." << endl;

	if (bytesRead > 0)
	{
		//Read the data
		int8_t* buffer = new int8_t[BUFFER_SIZE];
		int8_t* buffer_d = new int8_t[BUFFER_SIZE];
		int8_t* buffer_dd = new int8_t[BUFFER_SIZE];
		double* filtr_cpu = new double[31];
		double* filtr_gpu = new double[31];
		
		filtr_cpu = filter(wavHeader, 10000);

		hipMalloc((void**)&buffer_d, BUFFER_SIZE * sizeof(int8_t));
		hipMalloc((void**)&buffer_dd, BUFFER_SIZE * sizeof(int8_t));
		hipMalloc((void**)&filtr_gpu, 31 * sizeof(double));

		while ((bytesRead = fread(buffer, sizeof buffer[0], BUFFER_SIZE, wavFile)) > 0)
		{
			cout << "data bytes read: " << bytesRead << endl;
		}
		//allocate memory on GPU
		hipMemcpy(buffer_d, buffer, BUFFER_SIZE * sizeof(int8_t), hipMemcpyHostToDevice);
		hipMemcpy(filtr_gpu, filtr_cpu, 31 * sizeof(double), hipMemcpyHostToDevice);

		const int size_blocks = 1024;
		int num_blocks = BUFFER_SIZE / size_blocks - 1;
		//calling out GPU function(kernel)
		filterr <<<num_blocks, size_blocks >>> (buffer_d, buffer_dd, filtr_gpu);
		//transfering data from device to host
		hipMemcpy(buffer, buffer_dd, BUFFER_SIZE * sizeof(int8_t), hipMemcpyDeviceToHost);
		//writing calculated data to a file
		fwrite(buffer, wavHeader.Subchunk2Size, 1, output);

		hipFree(buffer_d);
		hipFree(buffer_dd);
		hipFree(filtr_gpu);
		
		
		delete[] buffer;
		buffer = nullptr;
		filelength = getFileSize(wavFile);

		cout << "File is                    :" << filelength << " bytes." << endl;
		cout << "RIFF header                :" << wavHeader.RIFF[0] << wavHeader.RIFF[1] << wavHeader.RIFF[2] << wavHeader.RIFF[3] << endl;
		cout << "WAVE header                :" << wavHeader.WAVE[0] << wavHeader.WAVE[1] << wavHeader.WAVE[2] << wavHeader.WAVE[3] << endl;
		cout << "FMT                        :" << wavHeader.fmt[0] << wavHeader.fmt[1] << wavHeader.fmt[2] << wavHeader.fmt[3] << endl;
		cout << "Data size                  :" << wavHeader.ChunkSize << endl;


		cout << "Sampling Rate              :" << wavHeader.SamplesPerSec << endl;
		cout << "Number of bits used        :" << wavHeader.bitsPerSample << endl;
		cout << "Number of channels         :" << wavHeader.NumOfChan << endl;
		cout << "Number of bytes per second :" << wavHeader.bytesPerSec << endl;
		cout << "Data length                :" << wavHeader.Subchunk2Size << endl;
		cout << "Audio Format               :" << wavHeader.AudioFormat << endl;


		cout << "Block align                :" << wavHeader.blockAlign << endl;
		cout << "Data string                :" << wavHeader.Subchunk2ID[0] << wavHeader.Subchunk2ID[1] << wavHeader.Subchunk2ID[2] << wavHeader.Subchunk2ID[3] << endl;

	}
	fclose(wavFile);
	fclose(output);

	return 0;
}

// find the file size
int getFileSize(FILE* inFile)
{
	int fileSize = 0;
	fseek(inFile, 0, SEEK_END);

	fileSize = ftell(inFile);

	fseek(inFile, 0, SEEK_SET);
	return fileSize;
}
//calculating lowpass filter impulse repsonse
double* filter(wav_hdr wavHeader, double limit_freq)
{
	double sampling_freq = wavHeader.SamplesPerSec;
	double usr_freq = limit_freq / sampling_freq / 2;
	int il_probek = 31;
	double* filtr = new double[il_probek];
	int n = 0;

	for(int i = -il_probek/2; i < 0; i++)
	{
		filtr[n] = sin(2 * 3.1415 * usr_freq * i) / (3.1415 * i);
		n++;
	}
	filtr[n] = 2 * usr_freq;
	n++;
	for (int j = 1; j <= il_probek / 2; j++)
	{
		filtr[n] = sin(2 * 3.1415 * usr_freq * j) / (3.1415 * j);
		n++;
	}
	return filtr;
}